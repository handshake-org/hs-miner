#include "hip/hip_runtime.h"
/*!
 * cuda.cu - CUDA Mining for hs-mminer
 * Copyright (c) 2019-2020, The Handshake Developers (MIT License).
 * https://github.com/handshake-org/hs-miner
 */

#include <stdio.h>
#include "common.h"
#include "blake2b.h"
#include "sha3.h"
#include "header.h"
#include "error.h"

typedef unsigned char BYTE;
typedef unsigned int  WORD;
typedef unsigned long long LONG;

#define BLAKE2B_ROUNDS 12
#define BLAKE2B_BLOCK_LENGTH 128
#define BLAKE2B_CHAIN_SIZE 8
#define BLAKE2B_CHAIN_LENGTH (BLAKE2B_CHAIN_SIZE * sizeof(int64_t))
#define BLAKE2B_STATE_SIZE 16
#define BLAKE2B_STATE_LENGTH (BLAKE2B_STATE_SIZE * sizeof(int64_t))

typedef struct {

    WORD digestlen;

    BYTE buff[BLAKE2B_BLOCK_LENGTH];
    int64_t chain[BLAKE2B_CHAIN_SIZE];
    int64_t state[BLAKE2B_STATE_SIZE];

    WORD pos;
    LONG t0;
    LONG t1;
    LONG f0;

} cuda_blake2b_ctx_t;

typedef cuda_blake2b_ctx_t CUDA_BLAKE2B_CTX;

__constant__ CUDA_BLAKE2B_CTX c_CTX;

__constant__ LONG BLAKE2B_IVS[8] = {
  0x6a09e667f3bcc908, 0xbb67ae8584caa73b, 0x3c6ef372fe94f82b, 0xa54ff53a5f1d36f1,
  0x510e527fade682d1, 0x9b05688c2b3e6c1f, 0x1f83d9abfb41bd6b, 0x5be0cd19137e2179
};

__constant__ unsigned char BLAKE2B_SIGMAS[12][16] = {
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
  { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
  { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
  { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
  { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
  { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
  { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
  { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
  { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
  { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
  { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 }
};

__device__ LONG cuda_blake2b_leuint64(BYTE *in)
{
  LONG a;
  memcpy(&a, in, 8);
  return a;
}

__device__ LONG cuda_blake2b_ROTR64(LONG a, BYTE b)
{
  return (a >> b) | (a << (64 - b));
}

__device__ void cuda_blake2b_G(cuda_blake2b_ctx_t *ctx, int64_t m1, int64_t m2, int32_t a, int32_t b, int32_t c, int32_t d)
{
    ctx->state[a] = ctx->state[a] + ctx->state[b] + m1;
    ctx->state[d] = cuda_blake2b_ROTR64(ctx->state[d] ^ ctx->state[a], 32);
    ctx->state[c] = ctx->state[c] + ctx->state[d];
    ctx->state[b] = cuda_blake2b_ROTR64(ctx->state[b] ^ ctx->state[c], 24);
    ctx->state[a] = ctx->state[a] + ctx->state[b] + m2;
    ctx->state[d] = cuda_blake2b_ROTR64(ctx->state[d] ^ ctx->state[a], 16);
    ctx->state[c] = ctx->state[c] + ctx->state[d];
    ctx->state[b] = cuda_blake2b_ROTR64(ctx->state[b] ^ ctx->state[c], 63);
}

__device__ __forceinline__ void cuda_blake2b_init_state(cuda_blake2b_ctx_t *ctx)
{

    memcpy(ctx->state, ctx->chain, BLAKE2B_CHAIN_LENGTH);

    // Set blake2b initialization vectors 0-3
    ctx->state[BLAKE2B_CHAIN_SIZE + 0] = 0x6a09e667f3bcc908;
    ctx->state[BLAKE2B_CHAIN_SIZE + 1] = 0xbb67ae8584caa73b;
    ctx->state[BLAKE2B_CHAIN_SIZE + 2] = 0x3c6ef372fe94f82b;
    ctx->state[BLAKE2B_CHAIN_SIZE + 3] = 0xa54ff53a5f1d36f1;

    // Hard code blake2b initialization vectors 4-7
    ctx->state[12] = ctx->t0 ^ 0x510e527fade682d1;
    ctx->state[13] = ctx->t1 ^ 0x9b05688c2b3e6c1f;
    ctx->state[14] = ctx->f0 ^ 0x1f83d9abfb41bd6b;
    ctx->state[15] = 0x5be0cd19137e2179;
}

__device__ __forceinline__ void cuda_blake2b_compress(cuda_blake2b_ctx_t *ctx, BYTE* in, WORD inoffset)
{
    cuda_blake2b_init_state(ctx);

    LONG  m[16] = {0};
#pragma unroll
    for (int j = 0; j < 16; j++)
        m[j] = cuda_blake2b_leuint64(in + inoffset + (j << 3));

    // 12 blake2b rounds in total
    // round 0
    cuda_blake2b_G(ctx, m[0], m[1], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[2], m[3], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[4], m[5], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[6], m[7], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[8], m[9], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[10], m[11], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[12], m[13], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[14], m[15], 3, 4, 9, 14);

    // round 1
    cuda_blake2b_G(ctx, m[14], m[10], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[4], m[8], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[9], m[15], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[13], m[6], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[1], m[12], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[0], m[2], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[11], m[7], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[5], m[3], 3, 4, 9, 14);


    // round 2
    cuda_blake2b_G(ctx, m[11], m[8], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[12], m[0], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[5], m[2], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[15], m[13], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[10], m[14], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[3], m[6], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[7], m[1], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[9], m[4], 3, 4, 9, 14);

    // round 3
    cuda_blake2b_G(ctx, m[7], m[9], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[3], m[1], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[13], m[12], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[11], m[14], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[2], m[6], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[5], m[10], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[4], m[0], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[15], m[8], 3, 4, 9, 14);

    // round 4
    cuda_blake2b_G(ctx, m[9], m[0], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[5], m[7], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[2], m[4], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[10], m[15], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[14], m[1], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[11], m[12], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[6], m[8], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[3], m[13], 3, 4, 9, 14);

    // round 5
    cuda_blake2b_G(ctx, m[2], m[12], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[6], m[10], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[0], m[11], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[8], m[3], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[4], m[13], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[7], m[5], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[15], m[14], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[1], m[9], 3, 4, 9, 14);


    // round 6
    cuda_blake2b_G(ctx, m[12], m[5], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[1], m[15], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[14], m[13], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[4], m[10], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[0], m[7], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[6], m[3], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[9], m[2], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[8], m[11], 3, 4, 9, 14);

    // round 7
    cuda_blake2b_G(ctx, m[13], m[11], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[7], m[14], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[12], m[1], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[3], m[9], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[5], m[0], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[15], m[4], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[8], m[6], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[2], m[10], 3, 4, 9, 14);

    // round 8
    cuda_blake2b_G(ctx, m[6], m[15], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[14], m[9], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[11], m[3], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[0], m[8], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[12], m[2], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[13], m[7], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[1], m[4], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[10], m[5], 3, 4, 9, 14);

    // round 9
    cuda_blake2b_G(ctx, m[10], m[2], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[8], m[4], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[7], m[6], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[1], m[5], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[15], m[11], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[9], m[14], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[3], m[12], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[13], m[0], 3, 4, 9, 14);

    // round 10
    cuda_blake2b_G(ctx, m[0], m[1], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[2], m[3], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[4], m[5], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[6], m[7], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[8], m[9], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[10], m[11], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[12], m[13], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[14], m[15], 3, 4, 9, 14);

    // round 11
    cuda_blake2b_G(ctx, m[14], m[10], 0, 4, 8, 12);
    cuda_blake2b_G(ctx, m[4], m[8], 1, 5, 9, 13);
    cuda_blake2b_G(ctx, m[9], m[15], 2, 6, 10, 14);
    cuda_blake2b_G(ctx, m[13], m[6], 3, 7, 11, 15);
    cuda_blake2b_G(ctx, m[1], m[12], 0, 5, 10, 15);
    cuda_blake2b_G(ctx, m[0], m[2], 1, 6, 11, 12);
    cuda_blake2b_G(ctx, m[11], m[7], 2, 7, 8, 13);
    cuda_blake2b_G(ctx, m[5], m[3], 3, 4, 9, 14);

    for (int offset = 0; offset < BLAKE2B_CHAIN_SIZE; offset++)
        ctx->chain[offset] = ctx->chain[offset] ^ ctx->state[offset] ^ ctx->state[offset + 8];
}

__device__ void cuda_blake2b_init(cuda_blake2b_ctx_t *ctx, WORD digestbitlen)
{
    memset(ctx, 0, sizeof(cuda_blake2b_ctx_t));

    ctx->digestlen = digestbitlen >> 3;
    ctx->pos = 0;
    ctx->t0 = 0;
    ctx->t1 = 0;
    ctx->f0 = 0;

    // Inline the blake2b initialization vectors 0-7
    ctx->chain[0] = 0x6a09e667f3bcc908 ^ (ctx->digestlen | 0x1010000);
    ctx->chain[1] = 0xbb67ae8584caa73b;
    ctx->chain[2] = 0x3c6ef372fe94f82b;
    ctx->chain[3] = 0xa54ff53a5f1d36f1;
    ctx->chain[4] = 0x510e527fade682d1;
    ctx->chain[5] = 0x9b05688c2b3e6c1f;
    ctx->chain[6] = 0x1f83d9abfb41bd6b;
    ctx->chain[7] = 0x5be0cd19137e2179;
}

__device__ void cuda_blake2b_update(cuda_blake2b_ctx_t *ctx, BYTE* in, LONG inlen)
{
    if (inlen == 0)
        return;

    WORD start = 0;
    int64_t in_index = 0, block_index = 0;

    if (ctx->pos)
    {
        start = BLAKE2B_BLOCK_LENGTH - ctx->pos;
        if (start < inlen){
            memcpy(ctx->buff + ctx->pos, in, start);
            ctx->t0 += BLAKE2B_BLOCK_LENGTH;

            if (ctx->t0 == 0) ctx->t1++;

            cuda_blake2b_compress(ctx, ctx->buff, 0);
            ctx->pos = 0;
            memset(ctx->buff, 0, BLAKE2B_BLOCK_LENGTH);
        } else {
            memcpy(ctx->buff + ctx->pos, in, inlen);//read the whole *in
            ctx->pos += inlen;
            return;
        }
    }

    block_index =  inlen - BLAKE2B_BLOCK_LENGTH;
    for (in_index = start; in_index < block_index; in_index += BLAKE2B_BLOCK_LENGTH)
    {
        ctx->t0 += BLAKE2B_BLOCK_LENGTH;
        if (ctx->t0 == 0)
            ctx->t1++;

        cuda_blake2b_compress(ctx, in, in_index);
    }

    memcpy(ctx->buff, in + in_index, inlen - in_index);
    ctx->pos += inlen - in_index;
}

__device__ void cuda_blake2b_final(cuda_blake2b_ctx_t *ctx, BYTE* out)
{
    ctx->f0 = 0xFFFFFFFFFFFFFFFFL;
    ctx->t0 += ctx->pos;
    if (ctx->pos > 0 && ctx->t0 == 0)
        ctx->t1++;

    cuda_blake2b_compress(ctx, ctx->buff, 0);
    memset(ctx->buff, 0, BLAKE2B_BLOCK_LENGTH);
    memset(ctx->state, 0, BLAKE2B_STATE_LENGTH);

    int i8 = 0;
    for (int i = 0; i < BLAKE2B_CHAIN_SIZE && ((i8 = i * 8) < ctx->digestlen); i++)
    {
        BYTE * BYTEs = (BYTE*)(&ctx->chain[i]);
        if (i8 < ctx->digestlen - 8)
            memcpy(out + i8, BYTEs, 8);
        else
            memcpy(out + i8, BYTEs, ctx->digestlen - i8);
    }
}

__global__ void kernel_blake2b_hash(BYTE* indata, WORD inlen, BYTE* outdata, WORD n_batch, WORD BLAKE2B_BLOCK_SIZE)
{
    WORD thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread >= n_batch)
    {
        return;
    }
    BYTE* in = indata  + thread * inlen;
    BYTE* out = outdata  + thread * BLAKE2B_BLOCK_SIZE;
    CUDA_BLAKE2B_CTX ctx = c_CTX;
    //if not precomputed CTX, call cuda_blake2b_init() with key
    cuda_blake2b_update(&ctx, in, inlen);
    cuda_blake2b_final(&ctx, out);
}

#define KECCAK_ROUND 24
#define KECCAK_STATE_SIZE 25
#define KECCAK_Q_SIZE 192

__constant__ LONG CUDA_KECCAK_CONSTS[24] = {
  0x0000000000000001, 0x0000000000008082, 0x800000000000808a, 0x8000000080008000,
  0x000000000000808b, 0x0000000080000001, 0x8000000080008081, 0x8000000000008009,
  0x000000000000008a, 0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
  0x000000008000808b, 0x800000000000008b, 0x8000000000008089, 0x8000000000008003,
  0x8000000000008002, 0x8000000000000080, 0x000000000000800a, 0x800000008000000a,
  0x8000000080008081, 0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};

typedef struct {

    BYTE sha3_flag;
    WORD digestbitlen;
    LONG rate_bits;
    LONG rate_BYTEs;
    LONG absorb_round;

    int64_t state[KECCAK_STATE_SIZE];
    BYTE q[KECCAK_Q_SIZE];

    LONG bits_in_queue;

} cuda_keccak_ctx_t;
typedef cuda_keccak_ctx_t CUDA_KECCAK_CTX;

__device__ LONG cuda_keccak_leuint64(void *in)
{
    LONG a;
    memcpy(&a, in, 8);
    return a;
}

__device__ int64_t cuda_keccak_MIN(int64_t a, int64_t b)
{
    if (a > b) return b;
    return a;
}

__device__ LONG cuda_keccak_UMIN(LONG a, LONG b)
{
    if (a > b) return b;
    return a;
}

__device__ void cuda_keccak_extract(cuda_keccak_ctx_t *ctx)
{
    LONG len = ctx->rate_bits >> 6;
    int64_t a;
    int s = sizeof(LONG);

    for (int i = 0;i < len;i++) {
        a = cuda_keccak_leuint64((int64_t*)&ctx->state[i]);
        memcpy(ctx->q + (i * s), &a, s);
    }
}

__device__ __forceinline__ LONG cuda_keccak_ROTL64(LONG a, LONG  b)
{
    return (a << b) | (a >> (64 - b));
}

__device__ void cuda_keccak_permutations(cuda_keccak_ctx_t * ctx)
{

    int64_t* A = ctx->state;;

    int64_t *a00 = A, *a01 = A + 1, *a02 = A + 2, *a03 = A + 3, *a04 = A + 4;
    int64_t *a05 = A + 5, *a06 = A + 6, *a07 = A + 7, *a08 = A + 8, *a09 = A + 9;
    int64_t *a10 = A + 10, *a11 = A + 11, *a12 = A + 12, *a13 = A + 13, *a14 = A + 14;
    int64_t *a15 = A + 15, *a16 = A + 16, *a17 = A + 17, *a18 = A + 18, *a19 = A + 19;
    int64_t *a20 = A + 20, *a21 = A + 21, *a22 = A + 22, *a23 = A + 23, *a24 = A + 24;

    for (int i = 0; i < KECCAK_ROUND; i++) {

        /* Theta */
        int64_t c0 = *a00 ^ *a05 ^ *a10 ^ *a15 ^ *a20;
        int64_t c1 = *a01 ^ *a06 ^ *a11 ^ *a16 ^ *a21;
        int64_t c2 = *a02 ^ *a07 ^ *a12 ^ *a17 ^ *a22;
        int64_t c3 = *a03 ^ *a08 ^ *a13 ^ *a18 ^ *a23;
        int64_t c4 = *a04 ^ *a09 ^ *a14 ^ *a19 ^ *a24;

        int64_t d1 = cuda_keccak_ROTL64(c1, 1) ^ c4;
        int64_t d2 = cuda_keccak_ROTL64(c2, 1) ^ c0;
        int64_t d3 = cuda_keccak_ROTL64(c3, 1) ^ c1;
        int64_t d4 = cuda_keccak_ROTL64(c4, 1) ^ c2;
        int64_t d0 = cuda_keccak_ROTL64(c0, 1) ^ c3;

        *a00 ^= d1;
        *a05 ^= d1;
        *a10 ^= d1;
        *a15 ^= d1;
        *a20 ^= d1;
        *a01 ^= d2;
        *a06 ^= d2;
        *a11 ^= d2;
        *a16 ^= d2;
        *a21 ^= d2;
        *a02 ^= d3;
        *a07 ^= d3;
        *a12 ^= d3;
        *a17 ^= d3;
        *a22 ^= d3;
        *a03 ^= d4;
        *a08 ^= d4;
        *a13 ^= d4;
        *a18 ^= d4;
        *a23 ^= d4;
        *a04 ^= d0;
        *a09 ^= d0;
        *a14 ^= d0;
        *a19 ^= d0;
        *a24 ^= d0;

        /* Rho pi */
        c1 = cuda_keccak_ROTL64(*a01, 1);
        *a01 = cuda_keccak_ROTL64(*a06, 44);
        *a06 = cuda_keccak_ROTL64(*a09, 20);
        *a09 = cuda_keccak_ROTL64(*a22, 61);
        *a22 = cuda_keccak_ROTL64(*a14, 39);
        *a14 = cuda_keccak_ROTL64(*a20, 18);
        *a20 = cuda_keccak_ROTL64(*a02, 62);
        *a02 = cuda_keccak_ROTL64(*a12, 43);
        *a12 = cuda_keccak_ROTL64(*a13, 25);
        *a13 = cuda_keccak_ROTL64(*a19, 8);
        *a19 = cuda_keccak_ROTL64(*a23, 56);
        *a23 = cuda_keccak_ROTL64(*a15, 41);
        *a15 = cuda_keccak_ROTL64(*a04, 27);
        *a04 = cuda_keccak_ROTL64(*a24, 14);
        *a24 = cuda_keccak_ROTL64(*a21, 2);
        *a21 = cuda_keccak_ROTL64(*a08, 55);
        *a08 = cuda_keccak_ROTL64(*a16, 45);
        *a16 = cuda_keccak_ROTL64(*a05, 36);
        *a05 = cuda_keccak_ROTL64(*a03, 28);
        *a03 = cuda_keccak_ROTL64(*a18, 21);
        *a18 = cuda_keccak_ROTL64(*a17, 15);
        *a17 = cuda_keccak_ROTL64(*a11, 10);
        *a11 = cuda_keccak_ROTL64(*a07, 6);
        *a07 = cuda_keccak_ROTL64(*a10, 3);
        *a10 = c1;

        /* Chi */
        c0 = *a00 ^ (~*a01 & *a02);
        c1 = *a01 ^ (~*a02 & *a03);
        *a02 ^= ~*a03 & *a04;
        *a03 ^= ~*a04 & *a00;
        *a04 ^= ~*a00 & *a01;
        *a00 = c0;
        *a01 = c1;

        c0 = *a05 ^ (~*a06 & *a07);
        c1 = *a06 ^ (~*a07 & *a08);
        *a07 ^= ~*a08 & *a09;
        *a08 ^= ~*a09 & *a05;
        *a09 ^= ~*a05 & *a06;
        *a05 = c0;
        *a06 = c1;

        c0 = *a10 ^ (~*a11 & *a12);
        c1 = *a11 ^ (~*a12 & *a13);
        *a12 ^= ~*a13 & *a14;
        *a13 ^= ~*a14 & *a10;
        *a14 ^= ~*a10 & *a11;
        *a10 = c0;
        *a11 = c1;

        c0 = *a15 ^ (~*a16 & *a17);
        c1 = *a16 ^ (~*a17 & *a18);
        *a17 ^= ~*a18 & *a19;
        *a18 ^= ~*a19 & *a15;
        *a19 ^= ~*a15 & *a16;
        *a15 = c0;
        *a16 = c1;

        c0 = *a20 ^ (~*a21 & *a22);
        c1 = *a21 ^ (~*a22 & *a23);
        *a22 ^= ~*a23 & *a24;
        *a23 ^= ~*a24 & *a20;
        *a24 ^= ~*a20 & *a21;
        *a20 = c0;
        *a21 = c1;

        /* Iota */
        *a00 ^= CUDA_KECCAK_CONSTS[i];
    }
}


__device__ void cuda_keccak_absorb(cuda_keccak_ctx_t *ctx, BYTE* in)
{

    LONG offset = 0;
    for (LONG i = 0; i < ctx->absorb_round; ++i) {
        ctx->state[i] ^= cuda_keccak_leuint64(in + offset);
        offset += 8;
    }

    cuda_keccak_permutations(ctx);
}

__device__ void cuda_keccak_pad(cuda_keccak_ctx_t *ctx)
{
    ctx->q[ctx->bits_in_queue >> 3] |= (1L << (ctx->bits_in_queue & 7));

    if (++(ctx->bits_in_queue) == ctx->rate_bits) {
        cuda_keccak_absorb(ctx, ctx->q);
        ctx->bits_in_queue = 0;
    }

    LONG full = ctx->bits_in_queue >> 6;
    LONG partial = ctx->bits_in_queue & 63;

    LONG offset = 0;
    for (int i = 0; i < full; ++i) {
        ctx->state[i] ^= cuda_keccak_leuint64(ctx->q + offset);
        offset += 8;
    }

    if (partial > 0) {
        LONG mask = (1L << partial) - 1;
        ctx->state[full] ^= cuda_keccak_leuint64(ctx->q + offset) & mask;
    }

    ctx->state[(ctx->rate_bits - 1) >> 6] ^= 9223372036854775808ULL;/* 1 << 63 */

    cuda_keccak_permutations(ctx);
    cuda_keccak_extract(ctx);

    ctx->bits_in_queue = ctx->rate_bits;
}

/*
 * Digestbitlen must be 128 224 256 288 384 512
 */
__device__ void cuda_keccak_init(cuda_keccak_ctx_t *ctx, WORD digestbitlen)
{
    memset(ctx, 0, sizeof(cuda_keccak_ctx_t));
    ctx->sha3_flag = 1;
    ctx->digestbitlen = digestbitlen;
    ctx->rate_bits = 1600 - ((ctx->digestbitlen) << 1);
    ctx->rate_BYTEs = ctx->rate_bits >> 3;
    ctx->absorb_round = ctx->rate_bits >> 6;
    ctx->bits_in_queue = 0;
}

/*
 * Digestbitlen must be 224 256 384 512
 */
__device__ void cuda_keccak_sha3_init(cuda_keccak_ctx_t *ctx, WORD digestbitlen)
{
    cuda_keccak_init(ctx, digestbitlen);
    ctx->sha3_flag = 1;
}

__device__ void cuda_keccak_update(cuda_keccak_ctx_t *ctx, BYTE *in, LONG inlen)
{
    int64_t BYTEs = ctx->bits_in_queue >> 3;
    int64_t count = 0;
    while (count < inlen) {
        if (BYTEs == 0 && count <= ((int64_t)(inlen - ctx->rate_BYTEs))) {
            do {
                cuda_keccak_absorb(ctx, in + count);
                count += ctx->rate_BYTEs;
            } while (count <= ((int64_t)(inlen - ctx->rate_BYTEs)));
        } else {
            int64_t partial = cuda_keccak_MIN(ctx->rate_BYTEs - BYTEs, inlen - count);
            memcpy(ctx->q + BYTEs, in + count, partial);

            BYTEs += partial;
            count += partial;

            if (BYTEs == ctx->rate_BYTEs) {
                cuda_keccak_absorb(ctx, ctx->q);
                BYTEs = 0;
            }
        }
    }
    ctx->bits_in_queue = BYTEs << 3;
}

__device__ void cuda_keccak_final(cuda_keccak_ctx_t *ctx, BYTE *out)
{
    if (ctx->sha3_flag) {
        int mask = (1 << 2) - 1;
        ctx->q[ctx->bits_in_queue >> 3] = (BYTE)(0x02 & mask);
        ctx->bits_in_queue += 2;
    }

    cuda_keccak_pad(ctx);
    LONG i = 0;

    while (i < ctx->digestbitlen) {
        if (ctx->bits_in_queue == 0) {
            cuda_keccak_permutations(ctx);
            cuda_keccak_extract(ctx);
            ctx->bits_in_queue = ctx->rate_bits;
        }

        LONG partial_block = cuda_keccak_UMIN(ctx->bits_in_queue, ctx->digestbitlen - i);
        memcpy(out + (i >> 3), ctx->q + (ctx->rate_BYTEs - (ctx->bits_in_queue >> 3)), partial_block >> 3);
        ctx->bits_in_queue -= partial_block;
        i += partial_block;
    }
}

__global__ void kernel_keccak_hash(BYTE* indata, WORD inlen, BYTE* outdata, WORD n_batch, WORD KECCAK_BLOCK_SIZE)
{
    WORD thread = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread >= n_batch)
    {
        return;
    }
    BYTE* in = indata  + thread * inlen;
    BYTE* out = outdata  + thread * KECCAK_BLOCK_SIZE;
    CUDA_KECCAK_CTX ctx;
    cuda_keccak_init(&ctx, KECCAK_BLOCK_SIZE << 3);
    cuda_keccak_update(&ctx, in, inlen);
    cuda_keccak_final(&ctx, out);
}


/**
 * The miner serialized header:
 *  nonce         - 4
 *  time          - 8
 *  padding       - 20
 *  prev_block    - 32
 *  tree_root     - 32
 *  mask hash     - 32
 *  extra_nonce   - 24
 *  reserved_root - 32
 *  witness_root  - 32
 *  merkle_root   - 32
 *  version       - 4
 *  bits          - 4
 */

// Global memory is underscore prefixed
__constant__ uint8_t _pre_header[96];
__constant__ uint8_t _target[32];
__constant__ uint8_t _padding[32];
__constant__ uint8_t _commit_hash[32];

__device__ int cuda_memcmp(const void *s1, const void *s2, size_t n) {
    const unsigned char *us1 = (const unsigned char *) s1;
    const unsigned char *us2 = (const unsigned char *) s2;
    while (n-- != 0) {
        if (*us1 != *us2) {
            return (*us1 < *us2) ? -1 : +1;
        }
        us1++;
        us2++;
    }
    return 0;
}

__global__ void kernel_hs_hash(
    uint32_t *out_nonce,
    bool *out_match,
    unsigned int start_nonce,
    unsigned int range,
    unsigned int threads
)
{
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= threads || tid >= range) {
        return;
    }

    // Set the nonce based on the start_nonce and thread.
    uint32_t nonce = start_nonce + tid;

    CUDA_BLAKE2B_CTX b_ctx;
    CUDA_KECCAK_CTX s_ctx;

    uint8_t hash[32];
    uint8_t left[64];
    uint8_t right[32];
    uint8_t share[128];

    // Create the share using the nonce,
    // pre_header and commit_hash.
    memcpy(share, &nonce, 4);
    memcpy(share + 4, _pre_header + 4, 92);
    memcpy(share + 96, _commit_hash, 32);

    // Generate left by hashing the share
    // with blake2b-512.
    cuda_blake2b_init(&b_ctx, 512);
    cuda_blake2b_update(&b_ctx, share, 128);
    cuda_blake2b_final(&b_ctx, left);

    // Generate right by hashing the share
    // and first 8 bytes of padding with
    // sha3-256.
    cuda_keccak_init(&s_ctx, 256);
    cuda_keccak_update(&s_ctx, share, 128);
    cuda_keccak_update(&s_ctx, _padding, 8);
    cuda_keccak_final(&s_ctx, right);

    // Generate share hash by hashing together
    // the left, 32 bytes of padding and the
    // right with blake2b-256.
    cuda_blake2b_init(&b_ctx, 256);
    cuda_blake2b_update(&b_ctx, left, 64);
    cuda_blake2b_update(&b_ctx, _padding, 32);
    cuda_blake2b_update(&b_ctx, right, 32);
    cuda_blake2b_final(&b_ctx, hash);

    // Do a bytewise comparison to see if the
    // hash satisfies the target. This could be
    // either the network target or the pool target.
    if (cuda_memcmp(hash, _target, 32) <= 0) {
        *out_nonce = nonce;
        *out_match = true;
        return;
    }
}

// Calculate the commit hash on the CPU and copy to the GPU
// before starting the GPU kernel. This saves the need for each
// GPU thread to compute the exact same commit_hash.
void hs_commit_hash(const uint8_t *sub_header, const uint8_t *mask_hash)
{
    uint8_t sub_hash[32];
    uint8_t commit_hash[32];

    // Create the sub_hash by hashing the
    // sub_header with blake2b-256.
    hs_blake2b_ctx b_ctx;
    hs_blake2b_init(&b_ctx, 32);
    hs_blake2b_update(&b_ctx, sub_header, 128);
    hs_blake2b_final(&b_ctx, sub_hash, 32);

    // Create the commit_hash by hashing together
    // the sub_hash and the mask_hash with blake2b-256.
    // The mask_hash is included in the miner header serialization
    // that comes from `getwork` or stratum.
    hs_blake2b_init(&b_ctx, 32);
    hs_blake2b_update(&b_ctx, sub_hash, 32);
    hs_blake2b_update(&b_ctx, mask_hash, 32);
    hs_blake2b_final(&b_ctx, commit_hash, 32);

    hipMemcpyToSymbol(HIP_SYMBOL(_commit_hash), commit_hash, 32);
}

// At most 32 bytes of padding are needed, so calculate all 32
// bytes and then copy it to the GPU.
void hs_padding(const uint8_t *prev_block, const uint8_t *tree_root, size_t len)
{
    uint8_t padding[len];

    size_t i;
    for (i = 0; i < len; i++)
      padding[i] = prev_block[i % 32] ^ tree_root[i % 32];

    hipMemcpyToSymbol(HIP_SYMBOL(_padding), padding, 32);
}

// hs_miner_func for the cuda backend
int32_t hs_cuda_run(hs_options_t *options, uint32_t *result, uint8_t *extra_nonce, bool *match)
{
    uint32_t *out_nonce;
    bool *out_match;

    hipSetDevice(options->device);
    hipMalloc(&out_nonce, sizeof(uint32_t));
    hipMalloc(&out_match, sizeof(bool));
    hipMemset(out_match, 0, sizeof(bool));

    // preheader + mask hash
    // nonce       - 4 bytes
    // time        - 8 bytes
    // pad         - 20 bytes
    // prev        - 32 bytes
    // tree root   - 32 bytes
    // mask hash   - 32 bytes
    // total       - 128 bytes

    // subheader
    // extra nonce - 24 bytes
    // reserved    - 32 bytes
    // witness     - 32 bytes
    // merkle      - 32 bytes
    // version     - 4 bytes
    // bits        - 4 bytes
    // total       - 128 bytes

    hipMemcpyToSymbol(HIP_SYMBOL(_pre_header), options->header, 96);
    hipMemcpyToSymbol(HIP_SYMBOL(_target), options->target, 32);

    // Pointers to prev block and tree root.
    hs_padding(options->header + 32, options->header + 64, 32);
    // Pointers to the subheader and mask hash
    hs_commit_hash(options->header + 128, options->header + 96);

    kernel_hs_hash<<<options->grids, options->blocks>>>(
        out_nonce,
        out_match,
        options->nonce,
        options->range,
        options->threads
    );
    hipMemcpy(result, out_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(match, out_match, sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(out_nonce);
    hipFree(out_match);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      printf("error hs cuda hash: %s \n", hipGetErrorString(error));
      return HS_ENOSOLUTION;
    }

    if (*match)
      return HS_SUCCESS;

    return HS_ENOSOLUTION;
}
